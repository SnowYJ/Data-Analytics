#include "hip/hip_runtime.h"
#ifndef __HIPCC__
#define __HIPCC__
#endif

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <math.h>
#include <omp.h>
#include "hip/hip_runtime.h"
#include ""

#include "NBody.h"
#include "NBodyVisualiser.h"
#include "hip/hip_texture_types.h"
#include "texture_fetch_functions.hpp"

#define USER_NAME "acz19yz"
#define BLOCK_SIZE 32

void print_help();

// command checker.
void command_check_NGM(int argc, char* argv[]);
void command_check_IF(int argc, char* argv[]);
void body_initializer(char* argv[]);

// strsep
char* strsep(char** stringp, const char* delim);
// error
void error_func();
// is integer
boolean is_int(char* str);

// serial
void serial_step(void);
void openMP_step(void);
void cuda_step(void);

void pointer_operation_start();
void pointer_operation_end();

// N, D, M and I.
int num; 
int grid; 
MODE mode; 
int iter = 1; 

// index of I or file_name.
int index_arg_f = 0;

// running visualizer if true.
boolean toggle = FALSE;
// command has argument -i or -f.
boolean cond_arg_i = FALSE;
boolean cond_arg_f = FALSE;

// host pointer.
nbody *body_pointer; 
nbody *temp_pointer;
float *acti_map;

// device variable.
__device__ __constant__ int nbody_count;
__device__ __constant__ int nbody_grid;

// device pointer.
nbody* d_body_pointer;
nbody* d_temp_pointer;

nbody_soa d_soa_body_pointer;
nbody_soa d_soa_temp_pointer;

float* d_acti_map;

// using texture memory.
texture<float, hipTextureType1D, hipReadModeElementType> arr_x;
texture<float, hipTextureType1D, hipReadModeElementType> arr_y;
texture<float, hipTextureType1D, hipReadModeElementType> arr_vx;
texture<float, hipTextureType1D, hipReadModeElementType> arr_vy;
texture<float, hipTextureType1D, hipReadModeElementType> arr_m;

// convert AoS to SoA for better bandwidth.
__global__ void AoS_to_SoA(nbody_soa d_soa_body_pointer, nbody* d_body_pointer) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	d_soa_body_pointer.x[i] = d_body_pointer[i].x;
	d_soa_body_pointer.y[i] = d_body_pointer[i].y;
	d_soa_body_pointer.vx[i] = d_body_pointer[i].vx;
	d_soa_body_pointer.vy[i] = d_body_pointer[i].vy;
	d_soa_body_pointer.m[i] = d_body_pointer[i].m;
}

// density array should be initialized every interation.
__global__ void cuda_reset_density(float* d_acti_map) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	d_acti_map[i] = 0.0f;
}

// force calculation function.
__global__ void cuda_step_texture_SoA(nbody_soa d_soa_temp_pointer, float* d_acti_map) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	float unit = 1.0f / nbody_grid;

	float body_x = tex1Dfetch(arr_x, i);
	float body_y = tex1Dfetch(arr_y, i);
	float body_vx = tex1Dfetch(arr_vx, i);
	float body_vy = tex1Dfetch(arr_vy, i);
	float body_m = tex1Dfetch(arr_m, i);

	float force_x = 0.0f;
	float force_y = 0.0f;

	// read from texture memory.
	for (int j = 0; j < nbody_count; j++) {
		if (j == i) continue;
		float buffer_x = tex1Dfetch(arr_x, j);
		float buffer_y = tex1Dfetch(arr_y, j);
		float buffer_m = tex1Dfetch(arr_m, j);
		float dis_x = buffer_x - body_x;
		float dis_y = buffer_y - body_y;
		float magnitude = (float)sqrt((double)dis_x * dis_x + (double)dis_y * dis_y);
		force_x += (buffer_m * dis_x) / (float)pow(((double)magnitude + (double)SOFTENING*SOFTENING), 3.0 / 2);
		force_y += (buffer_m * dis_y) / (float)pow(((double)magnitude + (double)SOFTENING*SOFTENING), 3.0 / 2);
	}
	force_x *= G * body_m;
	force_y *= G * body_m;

	float acc_x = force_x / body_m;
	float acc_y = force_y / body_m;

	float new_vx = body_vx + dt * acc_x;
	float new_vy = body_vy + dt * acc_y;
	float new_x = body_x + dt * new_vx;
	float new_y = body_y + dt * new_vy;

	// write to d_soa_temp_pointer in global memory.
	d_soa_temp_pointer.vx[i] = new_vx;
	d_soa_temp_pointer.vy[i] = new_vy;
	d_soa_temp_pointer.x[i] = new_x;
	d_soa_temp_pointer.y[i] = new_y;

	// update density.
	if (new_x > 0 && new_x < 1 && new_y > 0 && new_y < 1) {
		int index_y = (int) floor(new_y / unit);
		int index_x = (int) floor(new_x / unit);
		int index = nbody_grid * index_y + index_x;
		atomicAdd(&d_acti_map[index], (float) nbody_grid / nbody_count);
	}
}

void swap_pointer() {
	float* temp_x, * temp_y, * temp_vx, * temp_vy;

	temp_x = d_soa_temp_pointer.x;
	d_soa_temp_pointer.x = d_soa_body_pointer.x;
	d_soa_body_pointer.x = temp_x;

	temp_y = d_soa_temp_pointer.y;
	d_soa_temp_pointer.y = d_soa_body_pointer.y;
	d_soa_body_pointer.y = temp_y;

	temp_vx = d_soa_temp_pointer.vx;
	d_soa_temp_pointer.vx = d_soa_body_pointer.vx;
	d_soa_body_pointer.vx = temp_vx;

	temp_vy = d_soa_temp_pointer.vy;
	d_soa_temp_pointer.vy = d_soa_body_pointer.vy;
	d_soa_body_pointer.vy = temp_vy;
}

void cuda_step(void) {
	int num_block_simualtion = (int) ceil(num/(double)BLOCK_SIZE);
	int num_block_density = (int) ceil((grid*grid)/(double)BLOCK_SIZE);

	dim3 threadsPerBlock(BLOCK_SIZE, 1, 1);
	dim3 blocksPerGrid(num_block_simualtion, 1, 1);
	dim3 blocksPerGrid1(num_block_density, 1, 1);

	for (int i = 0; i < iter; i++) {
		// reset density.
		cuda_reset_density << <blocksPerGrid1, threadsPerBlock >> > (d_acti_map);
		hipDeviceSynchronize();
		// update temp nbody.
		cuda_step_texture_SoA << <blocksPerGrid, threadsPerBlock >> > (d_soa_temp_pointer, d_acti_map);
		hipDeviceSynchronize();
		// update d_body_pointer.
		swap_pointer();
	}
}

int main(int argc, char *argv[]) {
	
	// function: check command n, g, m.
	command_check_NGM(argc, argv);

	//  n, g, m initialization & allocate heap
	num = atoi(argv[1]);
	grid = atoi(argv[2]);
	if (strcmp(argv[3], "CPU") == 0) mode = CPU; 
	if (strcmp(argv[3], "OPENMP") == 0) mode = OPENMP; 
	if (strcmp(argv[3], "CUDA") == 0) mode = CUDA; 

	body_pointer = (struct nbody *) malloc(sizeof(struct nbody) * num);
	// used in serial and openmp step function.
	temp_pointer = (struct nbody*) malloc(sizeof(struct nbody) * num);
	acti_map = (float *)malloc(sizeof(float) * (grid * grid));

	// function: check command -i -f. 
	command_check_IF(argc,argv);

	// function: read file or random initialization 
	body_initializer(argv);

	// function: data move to device and texture memory bind.
	pointer_operation_start();

	// convert AoS to SoA.
	dim3 threadsPerBlock(BLOCK_SIZE, 1, 1);
	dim3 blocksPerGrid((int)ceil(num/(double)BLOCK_SIZE), 1, 1);
	AoS_to_SoA << < blocksPerGrid, threadsPerBlock >> > (d_soa_body_pointer, d_body_pointer);

	// running visualizer or iteration simulation
	if (toggle) {
		if (mode == 0) {
			initViewer(num, grid, mode, &serial_step);
			setNBodyPositions(body_pointer);
			setHistogramData(acti_map);
			startVisualisationLoop();
		}
		else if (mode == 1) {
			initViewer(num, grid, mode, &openMP_step);
			setNBodyPositions(body_pointer);
			setHistogramData(acti_map);
			startVisualisationLoop();
		}
		else {
			initViewer(num, grid, mode, &cuda_step);
			setNBodyPositions2f(d_soa_body_pointer.x, d_soa_body_pointer.y);
			setActivityMapData(d_acti_map);
			startVisualisationLoop();
		}
	}
	else {
		double begin, end, elapsed;

		begin = omp_get_wtime();
		if (mode == 0) {
			serial_step();
		}
		else if (mode == 1) {
			openMP_step();
		}
		else {
			cuda_step();
		}
		end = omp_get_wtime();
		elapsed = end - begin;
		int second = (int)elapsed;
		int millisecond = (int)(1000.0f * (elapsed - second));
		printf("Execution time %d seconds %d milliseconds. \n", second, millisecond);
	}

	// function: free and unbind pointers.
	pointer_operation_end();
	
	return 0;
}
// command check
void command_check_NGM(int argc, char* argv[]) {
	if (argc < 4 || argc > 8) {
		if (strcmp(argv[1], "-help") == 0) {
			print_help();
			exit(1);
		}
		else {
			fprintf(stderr, "Error command (using -help). \n");
			exit(1);
		}
	}
	// n, g, m. n and g should be positive int. m should be OPENMP or CPU.
	if (!atoi(argv[1]) || !atoi(argv[2]) || atoi(argv[1]) <= 0 || atoi(argv[2]) <= 0 || !is_int(argv[1]) || !is_int(argv[2])) {
		fprintf(stderr, "Error: the first two arguments must be positive integer. \n");
		exit(1);
	}
	if ((strcmp(argv[3], "CPU") != 0) && (strcmp(argv[3], "OPENMP") != 0) && (strcmp(argv[3], "CUDA") != 0)) {
		fprintf(stderr, "Error: the third argument should be CPU or OPENMP or CUDA. \n");
		exit(1);
	}
	// if argc > 4, but there is no -i or -f next.
	if (argc > 4) {
		if ((strcmp(argv[4], "-i") != 0 && strcmp(argv[4], "-f") != 0) && argc != 4) {
			fprintf(stderr, "Error command (using -help). \n");
			exit(1);
		}
	}
}
void command_check_IF(int argc, char* argv[]) {
	int index_arg_i;
	for (int i = 0; i < argc; i++) {
		// -f. -f and filename should appear toghther.
		if (strcmp(argv[i], "-f") == 0) {
			if (i == argc - 1) {
				fprintf(stderr, "can't use -f without file_name.");
				error_func();
			}
			if (strcmp(argv[i + 1], "-i") == 0) {
				fprintf(stderr, "can't use -f without file_name.");
				error_func();
			}
			if (argv[i + 1] != NULL) {
				cond_arg_f = TRUE;
				index_arg_f = i + 1;
			}
			else {
				cond_arg_f = FALSE;
			}
		}
		// -i. I should be positive int & -i I should appear together.
		if (strcmp(argv[i], "-i") == 0) {
			if (i == argc - 1) {
				fprintf(stderr, "Error: can't use -i without I.");
				error_func();
			}
			if (strcmp(argv[i + 1], "-f") == 0) {
				fprintf(stderr, "Error: can't use -i without I.");
				error_func();
			}
			if (atoi(argv[i + 1])) {
				if (atoi(argv[i + 1]) < 0 || !is_int(argv[i + 1])) {
					fprintf(stderr, "Error: I must be positive int.");
					error_func();
				}
				cond_arg_i = TRUE;
				index_arg_i = i + 1;
			}
			else {
				fprintf(stderr, "Error command (using -help). \n");
				error_func();
			}
		}
	}
	if ((cond_arg_i && !cond_arg_f) || (!cond_arg_i && cond_arg_f)) {
		if (argc != 6) {
			fprintf(stderr, "Error command (using -help). \n");
			error_func();
		}
	}
	// iteration or visualiser.
	if (cond_arg_i) {
		iter = atoi(argv[index_arg_i]);
	}
	else {
		toggle = TRUE;
	}
}
void body_initializer(char* argv[]) {
	FILE* f = NULL;
	int k = 0; // index of heap.
	if (cond_arg_f) {
		f = fopen(argv[index_arg_f], "r");
		if (f == NULL) {
			fprintf(stderr, "Error: don't find file! \n");
			error_func();
		}
		// read format data from file.
		char str[30];
		while (fgets(str, 30, f) != NULL) {
			// avoid blank or comment.
			if (str[0] == '#' || str[0] == '\n' || str[0] == '\r') {
				continue;
			}
			else {
				if (k == num) {
					fprintf(stderr, "Error: file is bigger than allocated heap. \n");
					fclose(f);
					error_func();
				}
				char* ptr;
				char* tok;
				ptr = str;
				int index = 0;
				while ((tok = strsep(&ptr, ",")) != NULL) {
					if (strlen(tok) > 1) {
						// read from file.
						if (index == 0) body_pointer[k].x = (float)atof(tok);
						else if (index == 1) body_pointer[k].y = (float)atof(tok);
						else if (index == 2) body_pointer[k].vx = (float)atof(tok);
						else if (index == 3) body_pointer[k].vy = (float)atof(tok);
						else if (index == 4) body_pointer[k].m = (float)atof(tok);
						else fprintf(stderr, "Error: less than 5. \n");
					}
					else {
						// random.
						if (index == 0) body_pointer[k].x = rand() / (RAND_MAX + 1.0f);
						else if (index == 1) body_pointer[k].y = rand() / (RAND_MAX + 1.0f);
						else if (index == 2) body_pointer[k].vx = 0.0f;
						else if (index == 3) body_pointer[k].vy = 0.0f;
						else if (index == 4) body_pointer[k].m = 1.0f / num;
						else fprintf(stderr, "Error: less than 5. \n");
					}
					index++;
				}
				k++;
				index = 0;
			}
		}
		fclose(f);
	}
	else {
		for (int j = 0; j < num; j++) {
			body_pointer[k].x = rand() / (RAND_MAX + 1.0f);
			body_pointer[k].y = rand() / (RAND_MAX + 1.0f);
			body_pointer[k].vx = 0;
			body_pointer[k].vy = 0;
			body_pointer[k].m = 1.0f / num;
			k++;
		}
	}
	if (num != k) {
		fprintf(stderr, "num is not equal to nbodies in file. \n");
		error_func();
	}
}

// step function.
void serial_step(void) {
	float unit = 1.0f / grid;
	for (int k = 0; k < iter; k++) {
		memcpy(temp_pointer, body_pointer, sizeof(nbody) * num);
		for (int i = 0; i < (grid * grid); i++) { 
			acti_map[i] = 0.0f; 
		}
		// when i == j, F is zero. 
		for (int i = 0; i < num; i++) {
			float force_x = 0.0f;
			float force_y = 0.0f;
			float body_x = temp_pointer[i].x;
			float body_y = temp_pointer[i].y;
			float body_m = temp_pointer[i].m;
			for (int j = 0; j < num; j++) {
				float dis_x = temp_pointer[j].x - body_x;
				float dis_y = temp_pointer[j].y - body_y;
				float magnitude = (float) sqrt((double) dis_x*dis_x + (double) dis_y*dis_y);
				force_x += (temp_pointer[j].m*dis_x) / (float) pow(((double) magnitude + (double) SOFTENING*SOFTENING), 3.0 / 2);
				force_y += (temp_pointer[j].m*dis_y) / (float) pow(((double) magnitude + (double) SOFTENING*SOFTENING), 3.0 / 2);
			}
			force_x *= G * body_m;
			force_y *= G * body_m;
			float acc_x = force_x / body_m;
			float acc_y = force_y / body_m;
			float new_x = body_x + dt * temp_pointer[i].vx;
			float new_y = body_y + dt * temp_pointer[i].vy;
			body_pointer[i].vx = temp_pointer[i].vx + dt * acc_x;
			body_pointer[i].vy = temp_pointer[i].vy + dt * acc_y;
			body_pointer[i].x = new_x;
			body_pointer[i].y = new_y;

			// update density.
			if (new_x < 0 || new_x >1 || new_x < 0 || new_y > 1) continue;
			int index_y = (int)floor(new_y / unit);
			int index_x = (int)floor(new_x / unit);
			int index = grid * index_y + index_x;
			acti_map[index] += (1.0f / num * 10.0f);
		}
	}
}
void openMP_step(void) {
	float unit = 1.0f / grid;
	omp_set_nested(1);
	int k;
	#pragma omp parallel for
	for (k = 0; k < iter; k++) {
		memcpy(temp_pointer, body_pointer, sizeof(nbody) * num);
		for (int i = 0; i < (grid * grid); i++) acti_map[i] = 0.0f;
		// when i == j, F is zero. 
		#pragma omp parallel default(none)
		{
			int i;
			#pragma omp for schedule(dynamic)
			for (i = 0; i < num; i++) {
				float force_x = 0.0f;
				float force_y = 0.0f;
				for (int j = 0; j < num; j++) {
					if (i == j) continue;
					float dis_x = temp_pointer[j].x - temp_pointer[i].x;
					float dis_y = temp_pointer[j].y - temp_pointer[i].y;
					float magnitude = (float)sqrt((double)dis_x * dis_x + (double)dis_y * dis_y);
					force_x += (temp_pointer[j].m * dis_x) / (float)pow(((double)magnitude + (double)SOFTENING * SOFTENING), 3.0 / 2);
					force_y += (temp_pointer[j].m * dis_y) / (float)pow(((double)magnitude + (double)SOFTENING * SOFTENING), 3.0 / 2);
				}

				force_x *= G * temp_pointer[i].m;
				force_y *= G * temp_pointer[i].m;
				float new_x = temp_pointer[i].x + dt * temp_pointer[i].vx;
				float new_y = temp_pointer[i].y + dt * temp_pointer[i].vy;
				body_pointer[i].vx = temp_pointer[i].vx + dt * (force_x / temp_pointer[i].m);
				body_pointer[i].vy = temp_pointer[i].vy + dt * (force_y / temp_pointer[i].m);
				body_pointer[i].x = new_x;
				body_pointer[i].y = new_y;

				// update density.
				if (new_x < 0 || new_x >1 || new_x < 0 || new_y > 1) continue;
				int index_y = (int)floor(new_y / unit);
				int index_x = (int)floor(new_x / unit);
				int index = grid * index_y + index_x;
				#pragma omp atomic
				acti_map[index] += (1.0f / num * 10.0f);
			}
		}
	}
}

void print_help(){
	printf("nbody_%s N D M [-i I] [-i input_file]\n", USER_NAME);

	printf("where:\n");
	printf("\tN                Is the number of bodies to simulate.\n");
	printf("\tD                Is the integer dimension of the activity grid. The Grid has D*D locations.\n");
	printf("\tM                Is the operation mode, either  'CPU' or 'OPENMP'\n");
	printf("\t[-i I]           Optionally specifies the number of simulation iterations 'I' to perform. Specifying no value will use visualisation mode. \n");
	printf("\t[-f input_file]  Optionally specifies an input file with an initial N bodies of data. If not specified random data will be created.\n");
}
void error_func() {
	free(body_pointer);
	free(acti_map);
	exit(1);
}
char* strsep(char** stringp, const char* delim) {
	char* s;
	const char* spanp;
	int c, sc;
	char* tok;

	if ((s = *stringp) == NULL)
		return (NULL);
	for (tok = s;;) {
		c = *s++;
		spanp = delim;
		do {
			if ((sc = *spanp++) == c) {
				if (c == 0)
					s = NULL;
				else
					s[-1] = 0;
				*stringp = s;
				return (tok);
			}
		} while (sc != 0);
	}
	/* NOTREACHED */
}
boolean is_int(char* str) {
	boolean is_int = TRUE;
	for (int i = 0; i < (signed int) strlen(str); i++) {
		if (str[i] == '.') is_int = FALSE;
	}
	return is_int;
}

void pointer_operation_start() {
	// allocate device memory.
	hipMalloc((void**)&d_body_pointer, sizeof(struct nbody) * num);
	hipMalloc((void**)&d_temp_pointer, sizeof(struct nbody) * num);
	hipMalloc((void**)&d_acti_map, sizeof(float) * (grid * grid));

	// copy data from host to device. nbody, num, grid.
	hipMemcpy(d_body_pointer, body_pointer, sizeof(struct nbody) * num, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(nbody_count), &num, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(nbody_grid), &grid, sizeof(int));

	// d_soa_body_pointer will not be update within force calculation.
	hipMalloc((void**)&d_soa_body_pointer.x, sizeof(float) * num);
	hipMalloc((void**)&d_soa_body_pointer.y, sizeof(float) * num);
	hipMalloc((void**)&d_soa_body_pointer.vx, sizeof(float) * num);
	hipMalloc((void**)&d_soa_body_pointer.vy, sizeof(float) * num);
	hipMalloc((void**)&d_soa_body_pointer.m, sizeof(float) * num);

	// d_soa_temp_pointer will be update within force calculation.
	hipMalloc((void**)&d_soa_temp_pointer.x, sizeof(float) * num);
	hipMalloc((void**)&d_soa_temp_pointer.y, sizeof(float) * num);
	hipMalloc((void**)&d_soa_temp_pointer.vx, sizeof(float) * num);
	hipMalloc((void**)&d_soa_temp_pointer.vy, sizeof(float) * num);
	hipMalloc((void**)&d_soa_temp_pointer.m, sizeof(float) * num);

	hipBindTexture(0, arr_x, d_soa_body_pointer.x, sizeof(float) * num);
	hipBindTexture(0, arr_y, d_soa_body_pointer.y, sizeof(float) * num);
	hipBindTexture(0, arr_vx, d_soa_body_pointer.vx, sizeof(float) * num);
	hipBindTexture(0, arr_vy, d_soa_body_pointer.vy, sizeof(float) * num);
	hipBindTexture(0, arr_m, d_soa_body_pointer.m, sizeof(float) * num);
}
void pointer_operation_end() {

	hipFree(d_body_pointer);
	hipFree(d_temp_pointer);
	hipFree(d_acti_map);

	hipUnbindTexture(arr_x);
	hipUnbindTexture(arr_y);
	hipUnbindTexture(arr_vx);
	hipUnbindTexture(arr_vy);
	hipUnbindTexture(arr_m);

	hipFree(d_soa_body_pointer.x);
	hipFree(d_soa_body_pointer.y);
	hipFree(d_soa_body_pointer.vx);
	hipFree(d_soa_body_pointer.vy);
	hipFree(d_soa_body_pointer.m);

	hipFree(d_soa_temp_pointer.x);
	hipFree(d_soa_temp_pointer.y);
	hipFree(d_soa_temp_pointer.vx);
	hipFree(d_soa_temp_pointer.vy);
	hipFree(d_soa_temp_pointer.m);

	free(body_pointer);
	free(temp_pointer);
	free(acti_map);
}